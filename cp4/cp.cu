#include <vector>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#define STEPS 16
using namespace std;

static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    cerr << "CUDA error: " << context << ": "
         << hipGetErrorString(err) << endl;
    exit(EXIT_FAILURE);
  }
}

#define CHECK(x) check(x, #x)

void normalize_rows(int ny, int nx, const float *data, float *normalized)
{
  for (int j = 0; j < ny; j++)
  {
    float mean = 0.0;
    float magnitude = 0.0;

    for (int i = 0; i < nx; i++)
      mean += data[i + j * nx] / nx;

    for (int i = 0; i < nx; i++)
    {
      normalized[i + j * nx] = (data[i + j * nx] - mean);
      magnitude += normalized[i + j * nx] * normalized[i + j * nx];
    }
    magnitude = sqrtf(magnitude);

    for (int i = 0; i < nx; i++)
    {
      normalized[i + j * nx] /= magnitude;
    }
  }
}

__global__ void calculate_result(int nx, int ny, float *result, float *normalized)
{
  int is = blockIdx.x * STEPS;
  int js = threadIdx.x * STEPS;

  if (is >= ny || js >= ny)
    return;

  for (int j = js; j < js + STEPS; j++)
    for (int i = is; i < is + STEPS; i++)
    {
      if (i >= ny || j >= ny)
        continue;

      float sum = 0.0;
      for (int k = 0; k < nx; ++k)
        sum += normalized[k + i * nx] * normalized[k + j * nx];

      result[i + j * ny] = sum;
    }
}

/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
void correlate(int ny, int nx, const float *data, float *result)
{
  int nnx = nx + STEPS - nx % STEPS;
  int nny = ny + STEPS - ny % STEPS;
  // float *normalized = (float *)malloc(nnx * nny * sizeof(float));
  float *normalized = (float *)calloc(nnx * nny, sizeof(float));
  normalize_rows(ny, nx, data, normalized);

  // Allocate memory & copy data to GPU
  float *normalizedGPU = NULL;
  CHECK(hipMalloc((void **)&normalizedGPU, nnx * nny * sizeof(float)));
  float *resultGPU = NULL;
  CHECK(hipMalloc((void **)&resultGPU, ny * ny * sizeof(float)));
  CHECK(hipMemcpy(normalizedGPU, normalized, nnx * nny * sizeof(float), hipMemcpyHostToDevice));

  calculate_result<<<nny / STEPS, nny / STEPS>>>(nx, ny, resultGPU, normalizedGPU);
  CHECK(hipGetLastError());

  CHECK(hipMemcpy(result, resultGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipFree(normalizedGPU));
  CHECK(hipFree(resultGPU));
}
