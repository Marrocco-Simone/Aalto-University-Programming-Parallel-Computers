#include <vector>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#define STEPS 16
using namespace std;

static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    cerr << "CUDA error: " << context << ": "
         << hipGetErrorString(err) << endl;
    exit(EXIT_FAILURE);
  }
}

#define CHECK(x) check(x, #x)

__global__ void normalize_rows(int ny, int nx, float *data, float *normalized)
{
  int j = blockIdx.x * STEPS + threadIdx.x;
  if (j >= ny)
    return;

  float mean = 0.0;
  float magnitude = 0.0;

  for (int i = 0; i < nx; i++)
    mean += data[i + j * nx] / nx;

  for (int i = 0; i < nx; i++)
  {
    normalized[i + j * nx] = (data[i + j * nx] - mean);
    magnitude += normalized[i + j * nx] * normalized[i + j * nx];
  }
  magnitude = sqrtf(magnitude);

  for (int i = 0; i < nx; i++)
  {
    normalized[i + j * nx] /= magnitude;
  }
}

__global__ void calculate_result(int nx, int ny, float *result, float *normalized)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= ny || j >= ny)
    return;
  if (i < j)
  {
    result[i + j * ny] = 0.0;
  }

  float sum = 0.0;
  for (int k = 0; k < nx; ++k)
    sum += normalized[k + i * nx] * normalized[k + j * nx];

  result[i + j * ny] = sum;
}

/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
void correlate(int ny, int nx, const float *data, float *result)
{
  int nnx = nx + STEPS - nx % STEPS;
  int nny = ny + STEPS - ny % STEPS;

  float *dataGPU = NULL;
  CHECK(hipMalloc((void **)&dataGPU, nx * ny * sizeof(float)));
  CHECK(hipMemcpy(dataGPU, data, nx * ny * sizeof(float), hipMemcpyHostToDevice));

  float *normalizedGPU = NULL;
  CHECK(hipMalloc((void **)&normalizedGPU, nnx * nny * sizeof(float)));
  normalize_rows<<<nny / STEPS, STEPS>>>(ny, nx, dataGPU, normalizedGPU);

  float *resultGPU = NULL;
  CHECK(hipMalloc((void **)&resultGPU, ny * ny * sizeof(float)));

  dim3 dimBlock(STEPS, STEPS);
  dim3 dimGrid(nny / STEPS, nny / STEPS);
  calculate_result<<<dimGrid, dimBlock>>>(nx, ny, resultGPU, normalizedGPU);
  CHECK(hipGetLastError());

  CHECK(hipMemcpy(result, resultGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipFree(dataGPU));
  CHECK(hipFree(normalizedGPU));
  CHECK(hipFree(resultGPU));
}
