#include <vector>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#define STEPS 64
using namespace std;

/*
  ! test command
  ./grading test-plain; ./grading test-asan; ./grading test-memcheck-initcheck --remote; ./grading benchmark --remote benchmarks/4b.txt
*/

static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    cerr << "CUDA error: " << context << ": "
         << hipGetErrorString(err) << endl;
    exit(EXIT_FAILURE);
  }
}

#define CHECK(x) check(x, #x)

__global__ void normalize_rows(int ny, int nx, int nnx, float *data, float *normalized)
{
  int j = blockIdx.x * STEPS + threadIdx.x;
  if (j >= ny)
    return;

  float mean = 0.0;
  float magnitude = 0.0;

  for (int i = 0; i < nx; i++)
    mean += data[i + j * nx] / nx;

  for (int i = 0; i < nx; i++)
  {
    normalized[i + j * nnx] = (data[i + j * nx] - mean);
    magnitude += normalized[i + j * nnx] * normalized[i + j * nnx];
  }
  magnitude = sqrtf(magnitude);

  for (int i = 0; i < nx; i++)
  {
    normalized[i + j * nnx] /= magnitude;
  }
}

__global__ void calculate_result(int nx, int nnx, int nny, float *result, float *normalized)
{
  int js = (threadIdx.x + blockIdx.x * blockDim.x) * 8;
  int is = (threadIdx.y + blockIdx.y * blockDim.y) * 8;

  if (is < js)
    return;

  for (int j = js; j < js + 8; j++)
    for (int i = is; i < is + 8; i++)
    {
      float sum = 0.0;
      for (int k = 0; k < nx; ++k)
        sum += normalized[k + i * nnx] * normalized[k + j * nnx];

      result[i + j * nny] = sum;
    }
}

/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
void correlate(int ny, int nx, const float *data, float *result)
{
  using std::chrono::duration;
  using std::chrono::duration_cast;
  using std::chrono::high_resolution_clock;
  using std::chrono::milliseconds;

  auto t1 = high_resolution_clock::now();

  int nnx = nx + STEPS - nx % STEPS;
  int nny = ny + STEPS - ny % STEPS;

  float *dataGPU = NULL;
  CHECK(hipMalloc((void **)&dataGPU, nx * ny * sizeof(float)));
  CHECK(hipMemcpy(dataGPU, data, nx * ny * sizeof(float), hipMemcpyHostToDevice));

  float *normalizedGPU = NULL;
  CHECK(hipMalloc((void **)&normalizedGPU, nnx * nny * sizeof(float)));
  CHECK(hipMemset(normalizedGPU, 0.0, nnx * nny * sizeof(float)));
  normalize_rows<<<nny / STEPS, STEPS>>>(ny, nx, nnx, dataGPU, normalizedGPU);

  float *resultGPU = NULL;
  CHECK(hipMalloc((void **)&resultGPU, nny * nny * sizeof(float)));
  CHECK(hipMemset(resultGPU, 0.0, nny * nny * sizeof(float)));

  auto t2 = high_resolution_clock::now();

  dim3 dimBlock(8, 8);
  dim3 dimGrid(nny / STEPS, nny / STEPS);
  calculate_result<<<dimGrid, dimBlock>>>(nx, nnx, nny, resultGPU, normalizedGPU);
  CHECK(hipGetLastError());

  float *result_padded = (float *)malloc(nny * nny * sizeof(float));
  CHECK(hipMemcpy(result_padded, resultGPU, nny * nny * sizeof(float), hipMemcpyDeviceToHost));
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < ny; i++)
      result[i + j * ny] = result_padded[i + j * nny];

  CHECK(hipFree(dataGPU));
  CHECK(hipFree(normalizedGPU));
  CHECK(hipFree(resultGPU));

  auto t3 = high_resolution_clock::now();

  printf("Initialization: %ld ms\n", duration_cast<milliseconds>(t2 - t1).count());
  printf("Main loop: %ld ms\n", duration_cast<milliseconds>(t3 - t2).count());
}
