#include <vector>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#define STEPS 64
using namespace std;

/*
  ! test command
  ./grading test-plain; ./grading test-asan; ./grading test-memcheck-initcheck --remote; ./grading benchmark --remote benchmarks/4b.txt
*/

static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    cerr << "CUDA error: " << context << ": "
         << hipGetErrorString(err) << endl;
    exit(EXIT_FAILURE);
  }
}

#define CHECK(x) check(x, #x)

__global__ void normalize_rows(int ny, int nx, float *data, float *normalized)
{
  int j = blockIdx.x * STEPS + threadIdx.x;
  if (j >= ny)
    return;

  float mean = 0.0;
  float magnitude = 0.0;

  for (int i = 0; i < nx; i++)
    mean += data[i + j * nx] / nx;

  for (int i = 0; i < nx; i++)
  {
    normalized[i + j * nx] = (data[i + j * nx] - mean);
    magnitude += normalized[i + j * nx] * normalized[i + j * nx];
  }
  magnitude = sqrtf(magnitude);

  for (int i = 0; i < nx; i++)
  {
    normalized[i + j * nx] /= magnitude;
  }
}

__global__ void calculate_result(int nx, int ny, float *result, float *normalized)
{
  int js = (threadIdx.x + blockIdx.x * blockDim.x) * 8;
  int is = (threadIdx.y + blockIdx.y * blockDim.y) * 8;
  int je = min(js + 8, ny);
  int ie = min(is + 8, ny);

  if (is < js)
    return;

  for (int j = js; j < je; j++)
    for (int i = is; i < ie; i++)
    {
      float sum = 0.0;
      for (int k = 0; k < nx; ++k)
        sum += normalized[k + i * nx] * normalized[k + j * nx];

      result[i + j * ny] = sum;
    }
}

/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
void correlate(int ny, int nx, const float *data, float *result)
{
  using std::chrono::duration;
  using std::chrono::duration_cast;
  using std::chrono::high_resolution_clock;
  using std::chrono::milliseconds;

  auto t1 = high_resolution_clock::now();

  int nnx = nx + STEPS - nx % STEPS;
  int nny = ny + STEPS - ny % STEPS;

  float *dataGPU = NULL;
  CHECK(hipMalloc((void **)&dataGPU, nx * ny * sizeof(float)));
  CHECK(hipMemcpy(dataGPU, data, nx * ny * sizeof(float), hipMemcpyHostToDevice));

  float *normalizedGPU = NULL;
  CHECK(hipMalloc((void **)&normalizedGPU, nnx * nny * sizeof(float)));
  normalize_rows<<<nny / STEPS, STEPS>>>(ny, nx, dataGPU, normalizedGPU);

  auto t2 = high_resolution_clock::now();

  float *resultGPU = NULL;
  CHECK(hipMalloc((void **)&resultGPU, ny * ny * sizeof(float)));
  CHECK(hipMemset(resultGPU, 0, ny * ny * sizeof(float)));

  dim3 dimBlock(8, 8);
  dim3 dimGrid(nny / STEPS, nny / STEPS);
  calculate_result<<<dimGrid, dimBlock>>>(nx, ny, resultGPU, normalizedGPU);
  CHECK(hipGetLastError());

  CHECK(hipMemcpy(result, resultGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipFree(dataGPU));
  CHECK(hipFree(normalizedGPU));
  CHECK(hipFree(resultGPU));

  auto t3 = high_resolution_clock::now();

  printf("Initialization: %ld ms\n", duration_cast<milliseconds>(t2 - t1).count());
  printf("Main loop: %ld ms\n", duration_cast<milliseconds>(t3 - t2).count());
}
