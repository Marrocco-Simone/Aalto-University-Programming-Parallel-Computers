#include <vector>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#define STEP 12
#define PADDING 144
using namespace std;

/*
  ! test command
  ./grading test-plain; ./grading test-asan; ./grading test-memcheck-initcheck --remote; ./grading benchmark-cache --remote benchmarks/4b.txt
*/

static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    cerr << "CUDA error: " << context << ": "
         << hipGetErrorString(err) << endl;
    exit(EXIT_FAILURE);
  }
}

#define CHECK(x) check(x, #x)

__global__ void normalize_rows(int ny, int nx, int nny, float *data, float *normalizedTransposed)
{
  int j = blockIdx.x * PADDING + threadIdx.x;
  if (j >= ny)
    return;

  float mean = 0.0;
  float magnitude = 0.0;

  for (int k = 0; k < nx; k++)
    mean += data[k + j * nx] / nx;

  for (int k = 0; k < nx; k++)
  {
    normalizedTransposed[j + k * nny] = (data[k + j * nx] - mean);
    magnitude += normalizedTransposed[j + k * nny] * normalizedTransposed[j + k * nny];
  }
  magnitude = sqrtf(magnitude);

  for (int k = 0; k < nx; k++)
  {
    normalizedTransposed[j + k * nny] /= magnitude;
  }
}

__global__ void calculate_result(int nx, int nny, float *result, float *normalizedTransposed)
{
  int is = (threadIdx.x + blockIdx.x * blockDim.x) * STEP;
  int js = (threadIdx.y + blockIdx.y * blockDim.y) * STEP;
  // printf("blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, threadIdx.y: %d, js: %d, is: %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, js, is);

  if (is < js)
    return;

  float sums[STEP][STEP] = {0.0};
  for (int k = 0; k < nx; ++k)
  {
    for (int j = js; j < js + STEP; j++)
      for (int i = is; i < is + STEP; i++)
        sums[i - is][j - js] += normalizedTransposed[i + k * nny] * normalizedTransposed[j + k * nny];
  }

  for (int j = js; j < js + STEP; j++)
    for (int i = is; i < is + STEP; i++)
      result[i + j * nny] = sums[i - is][j - js];
}

/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
void correlate(int ny, int nx, const float *data, float *result)
{
  using std::chrono::duration;
  using std::chrono::duration_cast;
  using std::chrono::high_resolution_clock;
  using std::chrono::milliseconds;

  auto t1 = high_resolution_clock::now();

  int nnx = nx + PADDING - nx % PADDING;
  int nny = ny + PADDING - ny % PADDING;

  float *dataGPU = NULL;
  CHECK(hipMalloc((void **)&dataGPU, nx * ny * sizeof(float)));
  CHECK(hipMemcpy(dataGPU, data, nx * ny * sizeof(float), hipMemcpyHostToDevice));

  float *normalizedTransposedGPU = NULL;
  CHECK(hipMalloc((void **)&normalizedTransposedGPU, nnx * nny * sizeof(float)));
  CHECK(hipMemset(normalizedTransposedGPU, 0, nnx * nny * sizeof(float)));
  normalize_rows<<<nny / PADDING, PADDING>>>(ny, nx, nny, dataGPU, normalizedTransposedGPU);

  float *resultGPU = NULL;
  CHECK(hipMalloc((void **)&resultGPU, nny * nny * sizeof(float)));
  CHECK(hipMemset(resultGPU, 0, nny * nny * sizeof(float)));

  auto t2 = high_resolution_clock::now();

  dim3 dimBlock(STEP, STEP);
  dim3 dimGrid(nny / PADDING, nny / PADDING);
  calculate_result<<<dimGrid, dimBlock>>>(nx, nny, resultGPU, normalizedTransposedGPU);
  CHECK(hipGetLastError());

  float *result_padded = (float *)malloc(nny * nny * sizeof(float));
  CHECK(hipMemcpy(result_padded, resultGPU, nny * nny * sizeof(float), hipMemcpyDeviceToHost));
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < ny; i++)
      result[i + j * ny] = result_padded[i + j * nny];

  free(result_padded);
  CHECK(hipFree(dataGPU));
  CHECK(hipFree(normalizedTransposedGPU));
  CHECK(hipFree(resultGPU));

  auto t3 = high_resolution_clock::now();

  printf("Initialization: %ld ms\n", duration_cast<milliseconds>(t2 - t1).count());
  printf("Main loop: %ld ms\n", duration_cast<milliseconds>(t3 - t2).count());
}
